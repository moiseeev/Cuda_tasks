#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

int toMultiple(int a, int b)
{
    int mod = a % b;
    if (mod != 0)
    {
        mod = b - mod;
        return a + mod;
    }
    return a;
}

int main()
{
    // Initialize cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int M = 3000;
    int N = 4500;
    int K = 6000;

    M = toMultiple(M, 16);
    N = toMultiple(N, 16);
    K = toMultiple(K, 16);

    float *h_A = new float[M * K];
    float *h_B = new float[K * N];
    float *h_C = new float[M * N];

    float *d_A, *d_B, *d_C;

    // Allocate memory on device
    hipMalloc((void **)&d_A, M * K * sizeof(float));
    hipMalloc((void **)&d_B, K * N * sizeof(float));
    hipMalloc((void **)&d_C, M * N * sizeof(float));

    // Initialize host data
    for (int i = 0; i < M * K; ++i)
    {
        h_A[i] = static_cast<float>(i);
    }

    for (int i = 0; i < K * N; ++i)
    {
        h_B[i] = static_cast<float>(i);
    }

    // Copy data to device
    hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice);

    float alpha = 1.0f;
    float beta = 0.0f;

    // Perform matrix multiplication
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result (for small matrices)
    if (M <= 16 && N <= 16)
    {
        for (int i = 0; i < M; ++i)
        {
            for (int j = 0; j < N; ++j)
            {
                std::cout << h_C[i * N + j] << " ";
            }
            std::cout << std::endl;
        }
    }

    // Free memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Destroy cuBLAS handle
    hipblasDestroy(handle);

    return 0;
}

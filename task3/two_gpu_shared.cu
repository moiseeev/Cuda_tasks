#include "hip/hip_runtime.h"
#include <iostream>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
#include <array>
#include <opencv2/opencv.hpp>

#define BLOCK_SIZE (32u)
#define FILTER_SIZE (9u)
#define TILE_SIZE (23u) // BLOCK_SIZE - 2( FILTER_SIZE/2)
#define SIGMA 2.0f

#define CUDA_CHECK_RETURN(value)                                  \
    {                                                             \
        hipError_t err = value;                                  \
        if (err != hipSuccess)                                   \
        {                                                         \
            fprintf(stderr, "Error %s at line %d in file %s\n",   \
                    hipGetErrorString(err), __LINE__, __FILE__); \
            exit(-1);                                             \
        }                                                         \
    }


__global__ void applyFilter(unsigned char *out, unsigned char *in,
                            unsigned int width, unsigned int height)
{
    int x_o = (TILE_SIZE * blockIdx.x) + threadIdx.x;
    int y_o = (TILE_SIZE * blockIdx.y) + threadIdx.y;

    int x_i = x_o - FILTER_SIZE / 2;
    int y_i = y_o - FILTER_SIZE / 2;

    __shared__ unsigned char sBuffer[BLOCK_SIZE][BLOCK_SIZE];

    if ((x_i >= 0) && (x_i < width) && (y_i >= 0) && (y_i < height))
    {
        sBuffer[threadIdx.y][threadIdx.x] = in[y_i * width + x_i];
    }
    else
    {
        sBuffer[threadIdx.y][threadIdx.x] = 0;
    }

    __syncthreads();

    int sum = 0;
    if ((threadIdx.x < TILE_SIZE) && (threadIdx.y < TILE_SIZE))
    {

        for (int r = 0; r < FILTER_SIZE; ++r)
        {
            for (int c = 0; c < FILTER_SIZE; ++c)
            {
                // float k_value = kernel[r * FILTER_SIZE + c];
                sum += sBuffer[threadIdx.y + r][threadIdx.x + c];
            }
        }
        sum = sum / (FILTER_SIZE * FILTER_SIZE);
        // write into the output
        if (x_o < width && y_o < height)
            out[y_o * width + x_o] = sum;
    }
}

void filterImageWithGPUs(
    unsigned char *inputImage, unsigned char *outputImage, int width, int height, int startY, int endY, int gpuId
)
{
    unsigned char *d_input, *d_output;

    // Adjust the startY and endY to include overlap (e.g. 1 pixel above and below)
    int overlap = FILTER_SIZE / 2; 
    int adjustedStartY = max(startY - overlap, 0);
    int adjustedEndY = min(endY + overlap, height);

    // Allocate memory on the device
    CUDA_CHECK_RETURN(hipSetDevice(gpuId));
    hipEvent_t start, stop;
    CUDA_CHECK_RETURN(hipEventCreate(&start));
    CUDA_CHECK_RETURN(hipEventCreate(&stop));

    CUDA_CHECK_RETURN(hipEventRecord(start));

    int totalHeight = adjustedEndY - adjustedStartY; // New height with overlap
    unsigned int d_size = width * totalHeight * sizeof(unsigned char);

    CUDA_CHECK_RETURN(hipMalloc(&d_input, d_size));
    CUDA_CHECK_RETURN(hipMalloc(&d_output, d_size));

    // Copy adjusted input image section to GPU (includes overlap)
    CUDA_CHECK_RETURN(hipMemcpy(d_input,
                                 inputImage + adjustedStartY * width, totalHeight * width * sizeof(unsigned char), hipMemcpyHostToDevice));

    // Launch the Gaussian blur kernel
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + TILE_SIZE - 1) / TILE_SIZE, (totalHeight + TILE_SIZE - 1) / TILE_SIZE);

    applyFilter<<<gridSize, blockSize>>>(d_output, d_input, width, totalHeight);

    CUDA_CHECK_RETURN(hipDeviceSynchronize())

    // Copy the output back to the host for the valid output section
    CUDA_CHECK_RETURN(hipMemcpy(outputImage + (startY * width),
                                 d_output + (startY - adjustedStartY) * width, (endY - startY) * width * sizeof(unsigned char), hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipEventRecord(stop));
    CUDA_CHECK_RETURN(hipEventSynchronize(stop));

    float milliseconds = 0;
    CUDA_CHECK_RETURN(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Время обработки половины канала: " << milliseconds << " мсек" << std::endl;
    // Free device memory
    CUDA_CHECK_RETURN(hipFree(d_input));
    CUDA_CHECK_RETURN(hipFree(d_output));
}

int main()
{
    std::cout << "Используемая память: shared memory" << std::endl;

    cv::Mat img = cv::imread("image.jpg", cv::IMREAD_COLOR);
    if (img.empty())
    {
        std::cerr << "Ошибка загрузки изображения!" << std::endl;
        return -1;
    }

    unsigned int width = img.cols;
    unsigned int height = img.rows;
    unsigned int size = width * height * sizeof(unsigned char);

    // результат фильтрации на хосте
    unsigned char *h_r_n = (unsigned char *)malloc(size);
    unsigned char *h_g_n = (unsigned char *)malloc(size);
    unsigned char *h_b_n = (unsigned char *)malloc(size);

    cv::Mat channels[3];
    cv::split(img, channels);

    int halfHeight = height / 2;

#pragma omp parallel sections
    {
#pragma omp section

        {
            // GPU 0
            std::cout << "Обработка верхней части изображения на GPU 0..." << std::endl;
            filterImageWithGPUs(channels[2].data, h_r_n, width, height, 0, halfHeight, 0);
            filterImageWithGPUs(channels[1].data, h_g_n, width, height, 0, halfHeight, 0);
            filterImageWithGPUs(channels[0].data, h_b_n, width, height, 0, halfHeight, 0);
            std::cout << "finished на GPU 0..." << std::endl;
        }

#pragma omp section
        {
            // GPU 1
            std::cout << "Обработка верхней части изображения на GPU 1..." << std::endl;
            filterImageWithGPUs(channels[2].data, h_r_n, width, height, halfHeight, height, 1);
            filterImageWithGPUs(channels[1].data, h_g_n, width, height, halfHeight, height, 1);
            filterImageWithGPUs(channels[0].data, h_b_n, width, height, halfHeight, height, 1);
            std::cout << "finished на GPU 1..." << std::endl;
        }
    }

    cv::Mat output_img(height, width, CV_8UC3);
    for (int i = 0; i < height; ++i)
    {
        for (int j = 0; j < width; ++j)
        {
            output_img.at<cv::Vec3b>(i, j)[0] = h_b_n[i * width + j]; // B
            output_img.at<cv::Vec3b>(i, j)[1] = h_g_n[i * width + j]; // G
            output_img.at<cv::Vec3b>(i, j)[2] = h_r_n[i * width + j]; // R
        }
    }

    cv::imwrite("filtred_image.png", output_img);
    free(h_r_n);
    free(h_g_n);
    free(h_b_n);

    return 0;
}

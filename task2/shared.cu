#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
#include <array>
#include <opencv2/opencv.hpp>

#define BLOCK_SIZE (32u)
#define FILTER_SIZE (9u)
#define TILE_SIZE (23u) // BLOCK_SIZE - 2( FILTER_SIZE/2)
#define SIGMA 2.0f

#define CUDA_CHECK_RETURN(value)                                  \
    {                                                             \
        hipError_t err = value;                                  \
        if (err != hipSuccess)                                   \
        {                                                         \
            fprintf(stderr, "Error %s at line %d in file %s\n",   \
                    hipGetErrorString(err), __LINE__, __FILE__); \
            exit(-1);                                             \
        }                                                         \
    }


__global__ void applyFilter(unsigned char *out, unsigned char *in,
                            unsigned int width, unsigned int height)
{
    int x_o = (TILE_SIZE * blockIdx.x) + threadIdx.x;
    int y_o = (TILE_SIZE * blockIdx.y) + threadIdx.y;

    int x_i = x_o - FILTER_SIZE / 2;
    int y_i = y_o - FILTER_SIZE / 2;

    __shared__ unsigned char sBuffer[BLOCK_SIZE][BLOCK_SIZE];

    if ((x_i >= 0) && (x_i < width) && (y_i >= 0) && (y_i < height))
        sBuffer[threadIdx.y][threadIdx.x] = in[y_i * width + x_i];
    else
        sBuffer[threadIdx.y][threadIdx.x] = 0;

    __syncthreads();

    int sum = 0;
    if ((threadIdx.x < TILE_SIZE) && (threadIdx.y < TILE_SIZE))
    {

        for (int r = 0; r < FILTER_SIZE; ++r)
        {
            for (int c = 0; c < FILTER_SIZE; ++c)
            {
                sum += sBuffer[threadIdx.y + r][threadIdx.x + c];
            }
        }
        sum = sum / (FILTER_SIZE * FILTER_SIZE);
        // write into the output
        if (x_o < width && y_o < height)
            out[y_o * width + x_o] = sum;
    }
}

int main(int, char **)
{
    std::cout << "Используемая память: shared memory" << std::endl;

    cv::Mat img = cv::imread("image.jpg", cv::IMREAD_COLOR);

    unsigned int width = img.cols;
    unsigned int height = img.rows;

    unsigned int size = width * height * sizeof(unsigned char);

    // результат фильтрации на хосте
    unsigned char *h_r_n = (unsigned char *)malloc(size);
    unsigned char *h_g_n = (unsigned char *)malloc(size);
    unsigned char *h_b_n = (unsigned char *)malloc(size);

    cv::Mat channels[3];
    cv::split(img, channels);

    // результат фильтрации на устройстве
    unsigned char *d_r_n, *d_g_n, *d_b_n;
    CUDA_CHECK_RETURN(hipMalloc(&d_r_n, size));
    CUDA_CHECK_RETURN(hipMalloc(&d_g_n, size));
    CUDA_CHECK_RETURN(hipMalloc(&d_b_n, size));

    unsigned char *d_r, *d_g, *d_b;

    CUDA_CHECK_RETURN(hipMalloc(&d_r, size));
    CUDA_CHECK_RETURN(hipMalloc(&d_g, size));
    CUDA_CHECK_RETURN(hipMalloc(&d_b, size));

    CUDA_CHECK_RETURN(hipMemcpy(d_r, channels[2].data, size, hipMemcpyHostToDevice)); // R
    CUDA_CHECK_RETURN(hipMemcpy(d_g, channels[1].data, size, hipMemcpyHostToDevice)); // G
    CUDA_CHECK_RETURN(hipMemcpy(d_b, channels[0].data, size, hipMemcpyHostToDevice)); // B

    dim3 grid_size((width + TILE_SIZE - 1) / TILE_SIZE, (height + TILE_SIZE - 1) / TILE_SIZE);
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    applyFilter<<<grid_size, blockSize>>>(d_r_n, d_r, width, height);
    applyFilter<<<grid_size, blockSize>>>(d_g_n, d_g, width, height);
    applyFilter<<<grid_size, blockSize>>>(d_b_n, d_b, width, height);

    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    CUDA_CHECK_RETURN(hipMemcpy(h_r_n, d_r_n, size, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(h_g_n, d_g_n, size, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(h_b_n, d_b_n, size, hipMemcpyDeviceToHost));

    cv::Mat output_img(height, width, CV_8UC3);
    for (int i = 0; i < height; ++i)
    {
        for (int j = 0; j < width; ++j)
        {
            output_img.at<cv::Vec3b>(i, j)[0] = h_b_n[i * width + j]; // B
            output_img.at<cv::Vec3b>(i, j)[1] = h_g_n[i * width + j]; // G
            output_img.at<cv::Vec3b>(i, j)[2] = h_r_n[i * width + j]; // R
        }
    }

    cv::imwrite("filtred_image.png", output_img);

    free(h_r_n);
    free(h_g_n);
    free(h_b_n);
    hipFree(d_r_n);
    hipFree(d_g_n);
    hipFree(d_b_n);
    hipFree(d_r);
    hipFree(d_g);
    hipFree(d_b);

    std::cout << "Результат фильтрации: 'filtred_image.png'!" << std::endl;
    std::cout << "Время выполнения: " << milliseconds << " мсек" << std::endl; // # Время выполнения: 52.5158 мсек

    return 0;
}